#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#ifdef _WIN32
#include <conio.h>
#elif __linux__
#include <curses.h>
#endif

__global__ void what_is_my_id_2d(unsigned int * const block_x,
                              unsigned int * const block_y,
                              unsigned int * const thread,
                              unsigned int * const calc_thread,
                              unsigned int * const x_thread,
                              unsigned int * const y_thread,
                              unsigned int * const grid_dimx,
                              unsigned int * const block_dimx,
                              unsigned int * const grid_dimy,
                              unsigned int * const block_dimy)
{
    const unsigned int idx        = (blockIdx.x * blockDim.x) + threadIdx.x;   // Thread X index
    const unsigned int idy        = (blockIdx.y * blockDim.y) + threadIdx.y;   // Thread Y index
    const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;    // Thread absolute index

    block_x[thread_idx]     = blockIdx.x;
    block_y[thread_idx]     = blockIdx.y;
    thread[thread_idx]      = threadIdx.x;
    calc_thread[thread_idx] = thread_idx;
    x_thread[thread_idx]    = idx;
    y_thread[thread_idx]    = idy;
    grid_dimx[thread_idx]   = gridDim.x;
    block_dimx[thread_idx]  = blockDim.x;
    grid_dimy[thread_idx]   = gridDim.y;
    block_dimy[thread_idx]  = blockDim.y;

}

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE_X) * (ARRAY_SIZE_Y))

unsigned int cpu_block_x    [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y    [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread     [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_x_thread   [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_y_thread   [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx  [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy  [ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy [ARRAY_SIZE_Y][ARRAY_SIZE_X];


int main(void){

    // Dividing 32x16 matrix into blocks (2 options):
    dim3 threads_rect(32,4);
    dim3 blocks_rect(1,4);

    dim3 threads_square(16,8);
    dim3 blocks_square(2,2);

    // Declare pointers for GPU based params
    unsigned int* gpu_block_x;
    unsigned int* gpu_block_y;
    unsigned int* gpu_thread;
    unsigned int* gpu_calc_thread;
    unsigned int* gpu_x_thread;
    unsigned int* gpu_y_thread;
    unsigned int* gpu_grid_dimx;
    unsigned int* gpu_block_dimx;
    unsigned int* gpu_grid_dimy;
    unsigned int* gpu_block_dimy;

    // Allocate four arrays on GPU
    hipMalloc((void **)&gpu_block_x,       ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_y,       ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_thread,        ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread,   ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_x_thread,      ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_y_thread,      ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimx,     ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimx,    ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimy,     ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimy,    ARRAY_SIZE_IN_BYTES);

    // Execute kernel (one of them)
    // what_is_my_id_2d<<<blocks_rect, threads_rect>>>(gpu_block_x,
    what_is_my_id_2d<<<blocks_square, threads_square>>>(gpu_block_x,
                                                 gpu_block_y,
                                                 gpu_thread,
                                                 gpu_calc_thread,
                                                 gpu_x_thread,
                                                 gpu_y_thread,
                                                 gpu_grid_dimx,
                                                 gpu_block_dimx,
                                                 gpu_grid_dimy,
                                                 gpu_block_dimy);


    hipMemcpy(cpu_block_x,     gpu_block_x,        ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_y,     gpu_block_y,        ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_thread,      gpu_thread,         ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread,    ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_x_thread,    gpu_x_thread,       ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_y_thread,    gpu_y_thread,       ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_grid_dimx,   gpu_grid_dimx,      ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_dimx,  gpu_block_dimx,     ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_grid_dimy,   gpu_grid_dimy,      ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_dimy,  gpu_block_dimy,     ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);


    char ch;
    /* Iterate through the arrays and print */
    for (int y=0; y < ARRAY_SIZE_Y; y++)
    {
        for (int x=0; x < ARRAY_SIZE_X ; x++)
        {
            printf("CT: %2u BKX: %1u BKY: %1u TID: %2u YTID: %2u XTID: %2u GDX: %1u BDX: %1u GDY: %1u BDY: %1u\n",
            cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x],
            cpu_thread[y][x], cpu_y_thread[y][x], cpu_x_thread[y][x], cpu_grid_dimx[y][x],
            cpu_block_dimx[y][x], cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);
            /* Wait for any key so we can see the console window */
        }
        ch = getchar();
    }
    /* Wait for any key so we can see the console window */
    printf("Press any key to continue\n");

    ch = getchar();

    hipFree(gpu_block_x);
    hipFree(gpu_block_y);
    hipFree(gpu_thread);
    hipFree(gpu_calc_thread);
    hipFree(gpu_x_thread);
    hipFree(gpu_y_thread);
    hipFree(gpu_grid_dimx);
    hipFree(gpu_block_dimx);
    hipFree(gpu_grid_dimy);
    hipFree(gpu_block_dimy);
}
