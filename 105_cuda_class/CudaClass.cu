#include "hip/hip_runtime.h"
#include "CudaClass.cuh"

__global__ void Kernel (uint32_t *d_a, uint32_t *d_b, uint32_t *d_c){
        uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
        uint32_t tid = idx + idy * blockDim.x * gridDim.x;

        d_c[tid] = d_a[tid] + d_b[tid];
};

CudaClass::CudaClass(uint32_t *h_a, uint32_t *h_b, uint32_t *h_c, uint32_t length){

    this->length = length;
    this->mem_size = length * sizeof(uint32_t);

    this->h_a = h_a;
    this->h_b = h_b;
    this->h_c = h_c;

}

void CudaClass::AllocateDeviceMemory(){
    hipMalloc((void**) &d_a, mem_size);
    hipMalloc((void**) &d_b, mem_size);
    hipMalloc((void**) &d_c, mem_size);
}

void CudaClass::CopyInputToDevice(){
    hipMemcpy(d_a, h_a, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, mem_size, hipMemcpyHostToDevice);
}

void CudaClass::RunKernelGPU(){
    Kernel<<<length, 1>>>(d_a, d_b, d_c);
}

void CudaClass::CopyOutputToHost(){
    hipMemcpy(h_c, d_c, mem_size, hipMemcpyDeviceToHost);
}

void CudaClass::FreeDeviceMemory(){
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
