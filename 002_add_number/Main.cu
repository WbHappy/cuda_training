#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_number(int* a, int* b, int* c){
    *c = *a + *b;
}

int main(int argc, char **argv){
    int a, b, c;    // Host copies of variables
    int *d_a, *d_b, *d_c;   // Device copies of variables
    int size = sizeof(int);

    // Allocation of device's memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setting up variables on host
    a = 64;
    b = 641;

    // Copy inputs to device memory
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launching kernel on GPU
    add_number<<<1,1>>>(d_a, d_b, d_c);

    // Copy results back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("result: %d\n", c);

    return 0;
}
