#include "gpu_errchk.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if(code != hipSuccess)
    {
        fprintf(stderr, "GPU Assert: %s \nIn file: %s, line: %d \n", hipGetErrorString(code), file, line);
        if(abort) exit(code);
    }
}
