#include "hip/hip_runtime.h"
#include "gpu_errchk.cuh"

#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#define X 32
#define Y 32
#define SIZE (X*Y)

#define OFFSET_X 4
#define OFFSET_Y 4

#define IMG_X 256
#define IMG_Y 256
#define IMG_SIZE (IMG_X*IMG_Y)

#define THREADS SIZE
#define BLOCKS (IMG_SIZE/SIZE)

__global__ void shared_filling_pattern(uint8_t *hmap, uint8_t *cmap, uint8_t* tmap)
{
    __shared__ uint8_t hmap_shared[Y][X];

    int sidx = threadIdx.x * OFFSET_X;
    int sidy = threadIdx.y * OFFSET_Y;

    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * OFFSET_X;
    int idy = (blockIdx.y * blockDim.y + threadIdx.y) * OFFSET_Y;
    int tid = idy * gridDim.x * blockDim.x * OFFSET_X + idx;
    tmap[tid] = 255;

    int pid;

    for(int oy = 0; oy < OFFSET_Y; oy++)
    {
        for(int ox = 0; ox < OFFSET_X; ox++)
        {
            pid = (idy+oy) * gridDim.x * blockDim.x * OFFSET_X + (idx+ox);
            hmap_shared[sidy+oy][sidx+ox] = hmap[pid];
        }
    }


    if(blockIdx.x%2 == blockIdx.y%2)
    {
        for(int oy = 0; oy < OFFSET_Y; oy++)
        {
            for(int ox = 0; ox < OFFSET_X; ox++)
            {
                hmap_shared[sidy+oy][sidx+ox] = 255 - hmap_shared[sidy+oy][sidx+ox];
            }
        }
    }

    for(int oy = 0; oy < OFFSET_Y; oy++)
    {
        for(int ox = 0; ox < OFFSET_X; ox++)
        {
            pid = (idy+oy) * gridDim.x * blockDim.x * OFFSET_X + (idx+ox);
            cmap[pid] = hmap_shared[sidy+oy][sidx+ox];
        }
    }



}

int main(int argc, char const *argv[])
{

    cv::Mat hmap = cv::imread("hmap.png", cv::IMREAD_GRAYSCALE);
    cv::Mat cmap(hmap.rows, hmap.cols, hmap.type());
    cv::Mat tmap(hmap.rows, hmap.cols, hmap.type());
    for(int i = 0; i < hmap.rows * hmap.cols; i++){
        tmap.data[i] = 0;
    }
    uint8_t *dev_hmap, *dev_cmap, *dev_tmap;



    dim3 grid(IMG_X/X, IMG_Y/Y,1);
    dim3 block(X/OFFSET_X,Y/OFFSET_Y,1);

    gpuErrchk( hipMalloc((void**)&dev_hmap, hmap.rows * hmap.cols * sizeof(uint8_t)) );
    gpuErrchk( hipMalloc((void**)&dev_cmap, hmap.rows * hmap.cols * sizeof(uint8_t)) );
    gpuErrchk( hipMalloc((void**)&dev_tmap, hmap.rows * hmap.cols * sizeof(uint8_t)) );
    gpuErrchk( hipMemcpy(dev_hmap, hmap.data, hmap.rows * hmap.cols * sizeof(uint8_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dev_tmap, tmap.data, hmap.rows * hmap.cols * sizeof(uint8_t), hipMemcpyHostToDevice) );

    shared_filling_pattern<<< grid, block >>>(dev_hmap, dev_cmap, dev_tmap);

    gpuErrchk( hipMemcpy(cmap.data, dev_cmap, hmap.rows * hmap.cols * sizeof(uint8_t), hipMemcpyDeviceToHost) ) ;
    gpuErrchk( hipMemcpy(tmap.data, dev_tmap, hmap.rows * hmap.cols * sizeof(uint8_t), hipMemcpyDeviceToHost) ) ;

    gpuErrchk( hipFree(dev_hmap) );
    gpuErrchk( hipFree(dev_cmap) );



    cv::namedWindow("cmap", 0);
    cv::namedWindow("tmap", 0);
    cv::imshow("cmap", cmap);
    cv::imshow("tmap", tmap);
    cv::waitKey(0);

    return 0;
}
